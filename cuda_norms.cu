#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  cuda_norms.cu
 *
 *    Description:  File containing function source for cuda norm functions.
 *
 *        Version:  1.0
 *        Created:  17/02/16 11:46:13
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Michael Tierney (MT), tiernemi@tcd.ie
 *
 * =====================================================================================
 */

#include "cuda_norms.h"
#include "stdio.h"

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaReduceSumSqrt
 *    Arguments:  float * sumArray - Array of elements to be summed.
 *                int numElements - Number of elements in sumArray.
 *  Description:  Sums all terms in sumArray and gets its square root.
 * =====================================================================================
 */

__device__ float globSum = 0.f ;
__global__ void cudaReduceSumSqrt(float * sumArray, int numElements) {
	int id = threadIdx.x+blockIdx.x*blockDim.x ;
	if (id == 0) { 
		float threadSum = 0.f ;
		for (int i = 0 ; i < numElements ; ++i) {
			threadSum += sumArray[i] ;
		}
		globSum = sqrtf(threadSum) ;
	}
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaReduceMax
 *    Arguments:  float * maxArray - Array of elements to be reduced.
 *                int numElements - Number of elements in maxArray.
 *  Description:  Finds the largest value in maxArray and saves to globMax.
 * =====================================================================================
 */


__device__ float globMax = 0.f ;
__global__ void cudaReduceMax(float * maxArray, int numElements) {
	int id = threadIdx.x+blockIdx.x*blockDim.x ;
	if (id == 0) { 
		float threadMax = 0.f ;
		for (int i = 0 ; i < numElements ; ++i) {
			threadMax = threadMax < maxArray[i] ? maxArray[i] : threadMax ; 
		}
		globMax = threadMax ;
	}
}

/* 
* ===  FUNCTION  ===========================================================================
*         Name:  cudaCalcFrobeniusNormGPU
*    Arguments:  float * matDataGPU - Array containing matrix elements on GPU.
*                float * frobNormRow - Array containing the squared terms per row.
*                int numRows - Number of rows in matrix.
*                int numCols - Number of columns in matrix.
*  Description:  CUDA Global function for calculating max norm for the matrix. Calculates
*                max for each row. Reduction of rowMaxs array is carried out using another
*                function. Can be performed more efficiently using shared memory and atomic
*                functions.
* ==========================================================================================
*/

__global__ void cudaCalcFrobeniusNormGPU(float * matDataGPU, float * frobNormRow, int numRows, int numCols) {	
	int id = threadIdx.x+blockIdx.x*blockDim.x ;
	if (id < numCols) {
		frobNormRow[id] = 0.f ;
		for (int i = 0 ; i < numRows ; ++i) {
			frobNormRow[id] += matDataGPU[i*numCols+id]*matDataGPU[i*numCols+id] ;
		}
	}
}

/* 
 * ===  FUNCTION  ===========================================================================
 *         Name:  cudaCalcMaxNorm
 *    Arguments:  float * matDataGPU - Array containing matrix elements on GPU.
 *                float * rowMaxs - Array containing the maximum row entries on GPU.
 *                int numRows - Number of rows in matrix.
 *                int numCols - Number of columns in matrix.
 *  Description:  CUDA Global function for calculating max norm for the matrix. Calculates
 *                max for each row. Reduction of rowMaxs array is carried out using another
 *                function. Can be performed more efficiently using shared memory and atomic
 *                functions.
 * ==========================================================================================
 */

__global__ void cudaCalcMaxNorm(float * matDataGPU, float * colMaxs, int numRows, int numCols) {
	int colID = threadIdx.x+blockIdx.x*blockDim.x ;
	if (colID < numCols) {
		float max = 0.f ;
		for (int i = 0 ; i < numRows ; ++i) {
			float absVal = fabsf(matDataGPU[i*numCols + colID]) ;
			max = max < absVal ? absVal : max ;
		}
		colMaxs[colID] = max ;
	}
}		/* -----  end of function cudaCalcMaxNorm  ----- */

/* 
 * ===  FUNCTION  =========================================================================
 *         Name:  cudaCalcOneIndNorm
 *    Arguments:  float * matDataGPU - Array containing matrix elements on GPU.
 *                float * colNorms - Array containing the induced norm per column on GPU.
 *                int numRows - Number of rows in matrix.
 *                int numCols - Number of columns in matrix.
 *  Description:  CUDA Global function for calculating induced one-norm for the matrix.
 *                Calculates sum of absolute values of elements per column. The max is
 *                found by another function.
 * ========================================================================================
 */

__global__ void cudaCalcOneInducedNorm(float * matDataGPU, float * colNorms, int numRows, int numCols) {
	int colID = threadIdx.x+blockIdx.x*blockDim.x ;
	if (colID < numCols) {
		float sum = 0.f ;
		for (int i = 0 ; i < numRows ; ++i) {
			sum += fabsf(matDataGPU[i*numCols + colID]) ;
		}
		colNorms[colID] = sum ;
	}
}		/* -----  end of function cudaCalcOneIndNorm  ----- */

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaCalcInfIndNorm
  *    Arguments:  float * matDataGPU - Array containing matrix elements on GPU.
 *                float * rowNorms - Array containing the induced norm per row on GPU.
 *                int numRows - Number of rows in matrix.
 *                int numCols - Number of columns in matrix.
 *  Description:  CUDA Global function for calculating induced inf-norm for the matrix.
 *                Calculates sum of absolute values of elements per row. The max is
 *                found by another function.
 * =====================================================================================
 */

__global__ void cudaCalcInfInducedNorm(float * matDataGPU, float * rowNorms, int numRows, int numCols) {
	int rowID = threadIdx.x+blockIdx.x*blockDim.x ;
	if (rowID < numRows) {
		float sum = 0.f ;
		for (int i = 0 ; i < numCols ; ++i) {
			sum += fabsf(matDataGPU[rowID*numCols + i]) ;
		}
		rowNorms[rowID] = sum ;
	}
}		/* -----  end of function cudaCalcOneIndNorm  ----- */

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaUploadMatrixToGPU
 *    Arguments:  Matrix * mat - Matrix to upload to GPU.
 *                float ** matDataGPU - Array used to store matrix data on GPU.
 *  Description:  Uploads matrix to GPU using cuda memcopy and cuda malloc.
 * =====================================================================================
 */

void cudaUploadMatrixToGPU(Matrix * mat, float ** matDataGPUAdr) {
	hipError_t rt = hipMalloc((void **) matDataGPUAdr, sizeof(float)*mat->numRows*mat->numCols) ;
	if (rt != hipSuccess) {
		printf("hiiii\n");
	}
	hipError_t rt2 = hipMemcpy(*matDataGPUAdr, mat->data, sizeof(float)*mat->numRows*mat->numCols, hipMemcpyHostToDevice) ;
	if (rt != hipSuccess) {
		printf("hiiii2\n");
	}
}		/* -----  end of function cudaUploadMatrixToGPU  ----- */

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaFreeMatrixGPU
 *    Arguments:  Matrix * mat - Matrix that has been loaded to the GPU.
 *                float ** matDataGPU - Array used to store matrix data on GPU.
 *  Description:  Frees matrix data from GPU.
 * =====================================================================================
 */

void cudaFreeMatrixGPU(Matrix * mat, float * matDataGPU) {
	hipFree(matDataGPU) ;
}		/* -----  end of function cudaFreeMatrixGPU  ----- */

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaGetMaxNorm
 *    Arguments:  Matrix * mat - Matrix that we're finding the max norm of.
 *      Returns:  The max norm of the Matrix.
 *  Description:  This function finds the max norm of the matrix using CUDA. This norm 
 *                is simply the largest absolute element in the matrix.
 * =====================================================================================
 */

extern float cudaGetMaxNorm(Matrix * mat, int numThreads) {
	float * matDataGPU ;
	float * colMaxsGPU ;
	float max ;

	int block_size=numThreads ;
	dim3 dimBlock(block_size) ;
	dim3 dimGrid((mat->numCols/dimBlock.x) + (!(mat->numCols%dimBlock.x)?0:1) );

	cudaUploadMatrixToGPU(mat, &matDataGPU) ;
	hipMalloc((void **) &colMaxsGPU, sizeof(float)*mat->numCols) ;
	
	cudaCalcMaxNorm<<<dimGrid, dimBlock>>>(matDataGPU, colMaxsGPU, mat->numRows, mat->numCols) ;
	cudaReduceMax<<<1,1>>>(colMaxsGPU, mat->numCols) ;
	hipMemcpyFromSymbol(&max, HIP_SYMBOL(globMax), sizeof(float), 0, hipMemcpyDeviceToHost) ;

	cudaFreeMatrixGPU(mat, matDataGPU) ;
	hipFree(colMaxsGPU) ;

	return max ;
}		/* -----  end of function cudaGetMaxNorm  ----- */

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaGetFrobeniusNorm
 *    Arguments:  Matrix * mat - Matrix that we're finding the Frobenius norm of.
 *      Returns:  The Frobenius norm of the matrix.
 *  Description:  This function finds the Frobenius norm of the Matrix using CUDA. 
 *                The Frobenius norm is square root of the sum of the squares of each 
 *                element.
 * =====================================================================================
 */

extern float cudaGetFrobeniusNorm(Matrix * mat, int numThreads) {
	float * frobNormRow ;
	float * matDataGPU ;
	float frobNorm ;

	int N = mat->numCols ;
	int block_size=numThreads ;
	dim3 dimBlock(block_size) ;
	dim3 dimGrid((N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	cudaUploadMatrixToGPU(mat, &matDataGPU) ;
	hipMalloc((void **) &frobNormRow, sizeof(float)*N) ;
	// For each row square and sum. //
	cudaCalcFrobeniusNormGPU<<<dimGrid,dimBlock>>>(matDataGPU, frobNormRow, mat->numRows, mat->numCols) ;
	// Combine all row sums. //
	cudaReduceSumSqrt<<<1,1>>>(frobNormRow, N) ;
	// Get answer. //
	hipMemcpyFromSymbol(&frobNorm, HIP_SYMBOL(globSum), sizeof(float), 0, hipMemcpyDeviceToHost) ;

	cudaFreeMatrixGPU(mat, matDataGPU);
	hipFree(frobNormRow) ;
	return frobNorm ;
}		/* -----  end of function cudaGetMaxNorm  ----- */

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaGetOneInducedNorm    
 *    Arguments:  Matrix * mat - Matrix that we're finding the one-induced norm of.
 *      Returns:  The one-induced norm of the matrix.
 *  Description:  This function finds the one-induced norm of the Matrix using CUDA. 
 *                This norm is maximum of the values generated by summing the absolut
 *                values of the columns.
 * =====================================================================================
 */

extern float cudaGetOneInducedNorm(Matrix * mat, int numThreads) {
	float * oneIndNormArGPU ;
	float * matDataGPU ;
	float oneIndNorm ;

	int N =mat->numCols ;
	int block_size= numThreads ;
	dim3 dimBlock(block_size) ;
	dim3 dimGrid((N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	cudaUploadMatrixToGPU(mat, &matDataGPU) ;
	hipMalloc((void **) &oneIndNormArGPU, sizeof(float)*N) ;
	// For each column sum absolute value of elements. //
	cudaCalcOneInducedNorm<<<dimGrid,dimBlock>>>(matDataGPU, oneIndNormArGPU, mat->numRows, mat->numCols) ;
	// Find the largest sum. //
	cudaReduceMax<<<1,1>>>(oneIndNormArGPU, mat->numCols) ;
	hipMemcpyFromSymbol(&oneIndNorm, HIP_SYMBOL(globMax), sizeof(float), 0, hipMemcpyDeviceToHost) ;

	cudaFreeMatrixGPU(mat, matDataGPU);
	hipFree(oneIndNormArGPU) ;
	
	return oneIndNorm ;
}		/* -----  end of function cudaGetMaxNorm  ----- */

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaGetInfInducedNorm
 *    Arguments:  Matrix * mat - Matrix that we're finding the inf-induced norm of.
 *      Returns:  The inf-induced norm of the matrix.
 *  Description:  This function finds the inf-induced norm of the Matrix using CUDA. 
 *                This norm is maximum of the values generated by summing the absolute 
 *                values of the rows.
 * =====================================================================================
 */

extern float cudaGetInfInducedNorm(Matrix * mat, int numThreads) {
	float * infIndNormArGPU ;
	float * matDataGPU ;
	float infIndNorm ;

	int N =mat->numRows ;
	int block_size= numThreads ;
	dim3 dimBlock(block_size) ;
	dim3 dimGrid((N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	cudaUploadMatrixToGPU(mat, &matDataGPU) ;
	hipMalloc((void **) &infIndNormArGPU, sizeof(float)*N) ;
	// For each row sum absolute value of elements. //
	cudaCalcInfInducedNorm<<<dimGrid,dimBlock>>>(matDataGPU, infIndNormArGPU, mat->numRows, mat->numCols) ;
	// Find the largest sum. //
	cudaReduceMax<<<1,1>>>(infIndNormArGPU, mat->numCols) ;
	hipMemcpyFromSymbol(&infIndNorm, HIP_SYMBOL(globMax), sizeof(float), 0, hipMemcpyDeviceToHost) ;

	cudaFreeMatrixGPU(mat, matDataGPU);
	hipFree(infIndNormArGPU) ;
	
	return infIndNorm ;
}		/* -----  end of function cudaGetMaxNorm  ----- */

